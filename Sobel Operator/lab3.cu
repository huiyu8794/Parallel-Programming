
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <cassert>
#include <zlib.h>
#include <png.h>
// #include <stdlib.h>
// #include <assert.h>
// #include <math.h>
// #include <utils.h>

#define Z 2
#define Y 5
#define X 5
#define xBound X / 2
#define yBound Y / 2
#define SCALE 8

/* Hint 7 */
// device side can't call host function
// declare it to device function
__device__  inline int bound_check(int val, int lower, int upper) {
    if (val >= lower && val < upper)
        return 1;
    else
        return 0;
}


int read_png(const char* filename, unsigned char** image, unsigned* height, 
             unsigned* width, unsigned* channels) {

    unsigned char sig[8];
    FILE* infile;
    infile = fopen(filename, "rb");

    fread(sig, 1, 8, infile);
    if (!png_check_sig(sig, 8))
        return 1;   /* bad signature */

    png_structp png_ptr;
    png_infop info_ptr;

    png_ptr = png_create_read_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    if (!png_ptr)
        return 4;   /* out of memory */
  
    info_ptr = png_create_info_struct(png_ptr);
    if (!info_ptr) {
        png_destroy_read_struct(&png_ptr, NULL, NULL);
        return 4;   /* out of memory */
    }

    png_init_io(png_ptr, infile);
    png_set_sig_bytes(png_ptr, 8);
    png_read_info(png_ptr, info_ptr);
    int bit_depth, color_type;
    png_get_IHDR(png_ptr, info_ptr, width, height, &bit_depth, &color_type, NULL, NULL, NULL);

    png_uint_32  i, rowbytes;
    png_bytep  row_pointers[*height];
    png_read_update_info(png_ptr, info_ptr);
    rowbytes = png_get_rowbytes(png_ptr, info_ptr);
    *channels = (int) png_get_channels(png_ptr, info_ptr);

    if ((*image = (unsigned char *) malloc(rowbytes * *height)) == NULL) {
        png_destroy_read_struct(&png_ptr, &info_ptr, NULL);
        return 3;
    }

    for (i = 0;  i < *height;  ++i)
        row_pointers[i] = *image + i * rowbytes;
    png_read_image(png_ptr, row_pointers);
    png_read_end(png_ptr, NULL);
    return 0;
}

void write_png(const char* filename, png_bytep image, const unsigned height, const unsigned width, 
               const unsigned channels) {
    FILE* fp = fopen(filename, "wb");
    png_structp png_ptr = png_create_write_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    png_infop info_ptr = png_create_info_struct(png_ptr);
    png_init_io(png_ptr, fp);
    png_set_IHDR(png_ptr, info_ptr, width, height, 8,
                 PNG_COLOR_TYPE_RGB, PNG_INTERLACE_NONE,
                 PNG_COMPRESSION_TYPE_DEFAULT, PNG_FILTER_TYPE_DEFAULT);
    png_set_filter(png_ptr, 0, PNG_NO_FILTERS);
    png_write_info(png_ptr, info_ptr);
    png_set_compression_level(png_ptr, 1);

    png_bytep row_ptr[height];
    for (int i = 0; i < height; ++ i) {
        row_ptr[i] = image + i * width * channels * sizeof(unsigned char);
    }
    png_write_image(png_ptr, row_ptr);
    png_write_end(png_ptr, NULL);
    png_destroy_write_struct(&png_ptr, &info_ptr);
    fclose(fp);
}

__global__ void sobel_kernel(unsigned char *s, unsigned char *t, unsigned *height, unsigned *width,
           unsigned *channels) {
    /* Hint 4 */
    // get tid by blockIdx, blockDim threadIdx 
    // and replace y or x by tid
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid >= *height){
        // printf("tid %4d return\n", tid); 
        return;
    }

    /* Hint 5 */
    // use constant memory or shared memory for filter matrix
    // sync threads if necessary
    const char filter[Z][Y][X] = { { { -1, -4, -6, -4, -1 },
                                 { -2, -8, -12, -8, -2 },
                                 { 0, 0, 0, 0, 0 },
                                 { 2, 8, 12, 8, 2 },
                                 { 1, 4, 6, 4, 1 } },
                               { { -1, -2, 0, 2, 1 },
                                 { -4, -8, 0, 8, 4 },
                                 { -6, -12, 0, 12, 6 },
                                 { -4, -8, 0, 8, 4 },
                                 { -1, -2, 0, 2, 1 } } };
    double val[Z][3];
    int y = tid;
    // for (int y = tid; y < *height; y += gridDim.x * blockDim.x) {
        // printf("Thread %d, y = %4d, height = %4d\n", tid, y, *height);
        for (int x = 0; x < *width; ++x) {
            /* Z axis of filter */
            // printf("Thread %d, y = %4d, x = %4d\n", tid, y, x);
            for (int i = 0; i < Z; ++i) {
                
                val[i][2] = 0.;
                val[i][1] = 0.;
                val[i][0] = 0.;

                /* Y and X axis of filter */
                for (int v = -yBound; v <= yBound; ++v) {
                    for (int u = -xBound; u <= xBound; ++u) {
                        if (bound_check(x + u, 0, *width) && bound_check(y + v, 0, *height)) {
                            const unsigned char R =
                                s[*channels * (*width * (y + v) + (x + u)) + 2];
                            const unsigned char G =
                                s[*channels * (*width * (y + v) + (x + u)) + 1];
                            const unsigned char B =
                                s[*channels * (*width * (y + v) + (x + u)) + 0];
                            val[i][2] += R * filter[i][u + xBound][v + yBound];
                            val[i][1] += G * filter[i][u + xBound][v + yBound];
                            val[i][0] += B * filter[i][u + xBound][v + yBound]; 
                        }
                    }
                }
            }
            // printf("tid = %d Loop finished\n", tid);
            double totalR = 0.;
            double totalG = 0.;
            double totalB = 0.;
            for (int i = 0; i < Z; ++i) {
                totalR += val[i][2] * val[i][2];
                totalG += val[i][1] * val[i][1];
                totalB += val[i][0] * val[i][0];
            }
            totalR = sqrt(totalR) / SCALE;
            totalG = sqrt(totalG) / SCALE;
            totalB = sqrt(totalB) / SCALE;
            const unsigned char cR = (totalR > 255.) ? 255 : totalR;
            const unsigned char cG = (totalG > 255.) ? 255 : totalG;
            const unsigned char cB = (totalB > 255.) ? 255 : totalB;
            t[*channels * (*width * y + x) + 2] = cR;
            t[*channels * (*width * y + x) + 1] = cG;
            t[*channels * (*width * y + x) + 0] = cB;
        }
    // }
}

int main(int argc, char **argv) {
    assert(argc == 3);
    unsigned height, width, channels;
    unsigned *height_cuda, *width_cuda, *channels_cuda;
    unsigned char *src = NULL, *dst;
    unsigned char *src_cuda = NULL, *dst_cuda;

    /* read the image to src, and get height, width, channels */
    if (read_png(argv[1], &src, &height, &width, &channels)) {
        return -1;
    }

    dst = (unsigned char *)malloc(height * width * channels *
                                  sizeof(unsigned char));
    /* Hint 1 */
    // cudaMalloc(...) for device src and device dst
    hipMalloc((void **)&src_cuda, height * width * channels *sizeof(unsigned char));
    hipMalloc((void **)&dst_cuda, height * width * channels *sizeof(unsigned char));
    hipMalloc((void **)&height_cuda, sizeof(unsigned));
    hipMalloc((void **)&width_cuda, sizeof(unsigned));
    hipMalloc((void **)&channels_cuda, sizeof(unsigned));
    /* Hint 2 */
    // cudaMemcpy(...) copy source image to device (filter matrix if necessary)
    hipMemcpy(src_cuda, src, height * width * channels *sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(dst_cuda, dst, height * width * channels *sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(height_cuda, &height, sizeof(unsigned), hipMemcpyHostToDevice);
    hipMemcpy(width_cuda, &width, sizeof(unsigned), hipMemcpyHostToDevice);
    hipMemcpy(channels_cuda, &channels, sizeof(unsigned), hipMemcpyHostToDevice);;
    /* Hint 3 */
    // decide to use how many blocks and threads
    int threads_per_block = 100;
    sobel_kernel<<<height/threads_per_block + 1,threads_per_block>>>(src_cuda, dst_cuda, height_cuda, width_cuda, channels_cuda);
    // launch cuda kernel

    /* computation */
    
    // sobel(src, dst, height, width, channels);

    /* Hint 6 */
    // cudaMemcpy(...) copy result image to host
    hipMemcpy(dst, dst_cuda, height * width * channels *sizeof(unsigned char), hipMemcpyDeviceToHost);
    hipFree(src_cuda); hipFree(dst_cuda); hipFree(height_cuda); hipFree(width_cuda); hipFree(channels_cuda);
    write_png(argv[2], dst, height, width, channels);
    return 0;
}
